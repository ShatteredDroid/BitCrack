#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "KeySearchTypes.h"
#include "CudaKeySearchDevice.h"
#include "ptx.cuh"
#include "secp256k1.cuh"

#include "sha256.cuh"
#include "ripemd160.cuh"

#include "secp256k1.h"

#include "CudaHashLookup.cuh"
#include "CudaAtomicList.cuh"
#include "CudaDeviceKeys.cuh"

__constant__ unsigned int _INC_X[8];

__constant__ unsigned int _INC_Y[8];

__constant__ unsigned int *_CHAIN[1];

__constant__ unsigned int _INC_KEY[8];

__constant__ unsigned int *_PRIVATE_KEYS[1];

__constant__ unsigned int _NIBBLE_LIMIT;

__constant__ unsigned int _ITERATION_OFFSET[8];

static unsigned int *_chainBufferPtr = NULL;


__device__ void doRMD160FinalRound(const unsigned int hIn[5], unsigned int hOut[5])
{
    const unsigned int iv[5] = {
        0x67452301,
        0xefcdab89,
        0x98badcfe,
        0x10325476,
        0xc3d2e1f0
    };

    for(int i = 0; i < 5; i++) {
        hOut[i] = endian(hIn[i] + iv[(i + 1) % 5]);
    }
}


/**
 * Allocates device memory for storing the multiplication chain used in
 the batch inversion operation
 */
hipError_t allocateChainBuf(unsigned int count)
{
    hipError_t err = hipMalloc(&_chainBufferPtr, count * sizeof(unsigned int) * 8);

    if(err) {
        return err;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(_CHAIN), &_chainBufferPtr, sizeof(unsigned int *));
    if(err) {
        hipFree(_chainBufferPtr);
    }

    return err;
}

void cleanupChainBuf()
{
    if(_chainBufferPtr != NULL) {
        hipFree(_chainBufferPtr);
        _chainBufferPtr = NULL;
    }
}

/**
 *Sets the EC point which all points will be incremented by
 */
hipError_t setIncrementorPoint(const secp256k1::uint256 &x, const secp256k1::uint256 &y)
{
    unsigned int xWords[8];
    unsigned int yWords[8];

    x.exportWords(xWords, 8, secp256k1::uint256::BigEndian);
    y.exportWords(yWords, 8, secp256k1::uint256::BigEndian);

    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_INC_X), xWords, sizeof(unsigned int) * 8);
    if(err) {
        return err;
    }

    return hipMemcpyToSymbol(HIP_SYMBOL(_INC_Y), yWords, sizeof(unsigned int) * 8);
}

hipError_t setPrivateKeyIncrement(const secp256k1::uint256 &value)
{
    unsigned int words[8];
    value.exportWords(words, 8, secp256k1::uint256::BigEndian);

    return hipMemcpyToSymbol(HIP_SYMBOL(_INC_KEY), words, sizeof(unsigned int) * 8);
}

hipError_t setIterationOffset(const secp256k1::uint256 &value)
{
    unsigned int words[8];
    value.exportWords(words, 8, secp256k1::uint256::BigEndian);

    return hipMemcpyToSymbol(HIP_SYMBOL(_ITERATION_OFFSET), words, sizeof(unsigned int) * 8);
}

hipError_t setPrivateKeyBuffer(unsigned int *ptr)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(_PRIVATE_KEYS), &ptr, sizeof(unsigned int *));
}

hipError_t setNibbleLimit(unsigned int nibble)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(_NIBBLE_LIMIT), &nibble, sizeof(unsigned int));
}



__device__ void hashPublicKey(const unsigned int *x, const unsigned int *y, unsigned int *digestOut)
{
    unsigned int hash[8];

    sha256PublicKey(x, y, hash);

    // Swap to little-endian
    for(int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}

__device__ void hashPublicKeyCompressed(const unsigned int *x, unsigned int yParity, unsigned int *digestOut)
{
    unsigned int hash[8];

    sha256PublicKeyCompressed(x, yParity, hash);

    // Swap to little-endian
    for(int i = 0; i < 8; i++) {
        hash[i] = endian(hash[i]);
    }

    ripemd160sha256NoFinal(hash, digestOut);
}


__device__ __forceinline__ void addUint256(unsigned int value[8], const unsigned int addend[8])
{
    add_cc(value[7], value[7], addend[7]);
    addc_cc(value[6], value[6], addend[6]);
    addc_cc(value[5], value[5], addend[5]);
    addc_cc(value[4], value[4], addend[4]);
    addc_cc(value[3], value[3], addend[3]);
    addc_cc(value[2], value[2], addend[2]);
    addc_cc(value[1], value[1], addend[1]);
    addc(value[0], value[0], addend[0]);
}

__device__ __forceinline__ bool hasNibbleSequence(const unsigned int value[8], unsigned int nibbleLength)
{
    if(nibbleLength <= 1) {
        return false;
    }

    bool first = true;
    unsigned int prev = 0;
    unsigned int count = 0;

    for(int word = 0; word < 8; word++) {
        unsigned int w = value[word];

        for(int shift = 28; shift >= 0; shift -= 4) {
            unsigned int nibble = (w >> shift) & 0x0f;

            if(first) {
                prev = nibble;
                count = 1;
                first = false;
            } else if(nibble == prev) {
                count++;
                if(count >= nibbleLength) {
                    return true;
                }
            } else {
                prev = nibble;
                count = 1;
            }
        }
    }

    return false;
}


__device__ void setResultFound(int idx, bool compressed, unsigned int x[8], unsigned int y[8], unsigned int digest[5])
{
    CudaDeviceResult r;

    r.block = blockIdx.x;
    r.thread = threadIdx.x;
    r.idx = idx;
    r.compressed = compressed;

    for(int i = 0; i < 8; i++) {
        r.x[i] = x[i];
        r.y[i] = y[i];
    }

    doRMD160FinalRound(digest, r.digest);

    atomicListAdd(&r, sizeof(r));
}

__device__ void doIteration(int pointsPerThread, int compression)
{
    unsigned int *chain = _CHAIN[0];
    unsigned int *xPtr = ec::getXPtr();
    unsigned int *yPtr = ec::getYPtr();
    unsigned int *privPtr = _PRIVATE_KEYS[0];
    unsigned int nibbleLimit = _NIBBLE_LIMIT;
    bool useNibble = (nibbleLimit > 0) && (privPtr != NULL);

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0,0,0,0,0,0,0,1};
    for(int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];
        bool skip = false;

        if(useNibble) {
            unsigned int candidateKey[8];
            readInt(privPtr, i, candidateKey);
            addUint256(candidateKey, _ITERATION_OFFSET);
            skip = hasNibbleSequence(candidateKey, nibbleLimit);
        }

        readInt(xPtr, i, x);

        if(!skip) {
            unsigned int digest[5];

            if(compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
                unsigned int y[8];
                readInt(yPtr, i, y);

                hashPublicKey(x, y, digest);

                if(checkHash(digest)) {
                    setResultFound(i, false, x, y, digest);
                }
            }

            if(compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {
                hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

                if(checkHash(digest)) {
                    unsigned int y[8];
                    readInt(yPtr, i, y);
                    setResultFound(i, true, x, y, digest);
                }
            }
        }

        beginBatchAdd(_INC_X, x, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for(int i = pointsPerThread - 1; i >= 0; i--) {

        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAdd(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);

    }
}

__device__ void doIterationWithDouble(int pointsPerThread, int compression)
{
    unsigned int *chain = _CHAIN[0];
    unsigned int *xPtr = ec::getXPtr();
    unsigned int *yPtr = ec::getYPtr();
    unsigned int *privPtr = _PRIVATE_KEYS[0];
    unsigned int nibbleLimit = _NIBBLE_LIMIT;
    bool useNibble = (nibbleLimit > 0) && (privPtr != NULL);

    // Multiply together all (_Gx - x) and then invert
    unsigned int inverse[8] = {0,0,0,0,0,0,0,1};
    for(int i = 0; i < pointsPerThread; i++) {
        unsigned int x[8];
        bool skip = false;

        if(useNibble) {
            unsigned int candidateKey[8];
            readInt(privPtr, i, candidateKey);
            addUint256(candidateKey, _ITERATION_OFFSET);
            skip = hasNibbleSequence(candidateKey, nibbleLimit);
        }

        readInt(xPtr, i, x);

        if(!skip) {
            unsigned int digest[5];

            // uncompressed
            if(compression == PointCompressionType::UNCOMPRESSED || compression == PointCompressionType::BOTH) {
                unsigned int y[8];
                readInt(yPtr, i, y);
                hashPublicKey(x, y, digest);

                if(checkHash(digest)) {
                    setResultFound(i, false, x, y, digest);
                }
            }

            // compressed
            if(compression == PointCompressionType::COMPRESSED || compression == PointCompressionType::BOTH) {

                hashPublicKeyCompressed(x, readIntLSW(yPtr, i), digest);

                if(checkHash(digest)) {

                    unsigned int y[8];
                    readInt(yPtr, i, y);

                    setResultFound(i, true, x, y, digest);
                }
            }
        }

        beginBatchAddWithDouble(_INC_X, _INC_Y, xPtr, chain, i, i, inverse);
    }

    doBatchInverse(inverse);

    for(int i = pointsPerThread - 1; i >= 0; i--) {

        unsigned int newX[8];
        unsigned int newY[8];

        completeBatchAddWithDouble(_INC_X, _INC_Y, xPtr, yPtr, i, i, chain, inverse, newX, newY);

        writeInt(xPtr, i, newX);
        writeInt(yPtr, i, newY);

    }
}

/**
* Performs a single iteration
*/
__global__ void keyFinderKernel(int points, int compression)
{
    doIteration(points, compression);
}

__global__ void keyFinderKernelWithDouble(int points, int compression)
{
    doIterationWithDouble(points, compression);
}
